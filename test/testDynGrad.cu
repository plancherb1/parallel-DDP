#include "hip/hip_runtime.h"
/*******
nvcc -std=c++11 -o testDynGrad.exe testDynGrad.cu ../utils/cudaUtils.cu ../utils/threadUtils.cpp -gencode arch=compute_61,code=sm_61 -O3
*******/

#define PLANT 4
#define EE_COST_PDDP 0
#define MPC_MODE 1
#define FINITE_DIFF_EPSILON 0.001
#include "../config.cuh"
#include <random>
#define ERR_TOL 0.1 // percent error
#define RANDOM_MEAN 0
#define RANDOM_STDEVq 2
#define RANDOM_STDEVqd 5
#define RANDOM_STDEVu 50
std::default_random_engine randEng(time(0)); //seed
std::normal_distribution<double> randDistq(RANDOM_MEAN, RANDOM_STDEVq); //mean followed by stdiv
std::normal_distribution<double> randDistqd(RANDOM_MEAN, RANDOM_STDEVqd); //mean followed by stdiv
std::normal_distribution<double> randDistu(RANDOM_MEAN, RANDOM_STDEVu); //mean followed by stdiv

template <typename T>
__global__
void integratorGradientKernFiniteDiff(T *d_AB, T *d_x, T *d_u, T *d_I, T *d_Tbody, int ld_x, int ld_u, int ld_AB){
	__shared__ T s_x[2*STATE_SIZE_PDDP];
	__shared__ T s_u[2*CONTROL_SIZE];
	__shared__ T s_qdd[2*NUM_POS];
	for (int timestep = blockIdx.x; timestep < NUM_TIME_STEPS-1; timestep += gridDim.x){
		for (int outputCol = blockIdx.y; outputCol < STATE_SIZE_PDDP + CONTROL_SIZE; outputCol += gridDim.y){
			T *xk = &d_x[timestep*ld_x];
			T *uk = &d_u[timestep*ld_u];
			T *ABk = &d_AB[timestep*ld_AB*DIM_AB_c + ld_AB*outputCol];
			finiteDiffInner<T>(ABk,xk,uk,s_x,s_u,s_qdd,d_I,d_Tbody,outputCol);
		}
	}
}

template <typename T>
__host__
void integratorGradientFiniteDiff(T *h_AB, T *h_x, T *h_u, T *h_I, T *h_Tbody, int ld_x, int ld_u, int ld_AB){
	T s_x[2*STATE_SIZE_PDDP];
	T s_u[2*CONTROL_SIZE];
	T s_qdd[2*NUM_POS];
	for (int timestep = 0; timestep < NUM_TIME_STEPS-1; timestep++){
		for (int outputCol = 0; outputCol < STATE_SIZE_PDDP + CONTROL_SIZE; outputCol++){
			T *xk = &h_x[timestep*ld_x];
			T *uk = &h_u[timestep*ld_u];
			T *ABk = &h_AB[timestep*ld_AB*DIM_AB_c + ld_AB*outputCol];
			finiteDiffInner<T>(ABk,xk,uk,s_x,s_u,s_qdd,h_I,h_Tbody,outputCol);
		}
	}
}

template <typename T>
__global__
void integratorGradientKernAnalytic(T *d_AB, T *d_x, T *d_u, T *d_I, T *d_Tbody, int ld_x, int ld_u, int ld_AB){
	__shared__ T s_x[STATE_SIZE_PDDP];
	__shared__ T s_u[CONTROL_SIZE];
	__shared__ T s_qdd[NUM_POS];
	__shared__ T s_dqdd[3*NUM_POS*NUM_POS];
	T *xk = &d_x[blockIdx.x*ld_x];
	T *uk = &d_u[blockIdx.x*ld_u];
	T *ABk = &d_AB[blockIdx.x*ld_AB*DIM_AB_c];
	// load in the state and control
	#pragma unroll
	for (int ind = threadIdx.x + threadIdx.y*blockDim.x; ind < STATE_SIZE_PDDP; ind += blockDim.x*blockDim.y){
		s_x[ind] = xk[ind];      if (ind < CONTROL_SIZE){s_u[ind] = uk[ind];}
	}
	__syncthreads();
	// then compute the dynamics gradient
	_integratorGradient(ABk, s_x, s_u, s_qdd, s_dqdd, d_I, d_Tbody, (T)TIME_STEP, ld_AB);
}

template <typename T>
__host__
void integratorGradientAnalytic(T *h_AB, T *h_x, T *h_u, T *h_I, T *h_Tbody, int ld_x, int ld_u, int ld_AB){
	T s_qdd[NUM_POS];
	T s_dqdd[3*NUM_POS*NUM_POS];
	for (int k = 0; k < NUM_TIME_STEPS-1; k++){
		T *xk = &h_x[k*ld_x];
		T *uk = &h_u[k*ld_u];
		T *ABk = &h_AB[k*ld_AB*DIM_AB_c];
		_integratorGradient(ABk, xk, uk, s_qdd, s_dqdd, h_I, h_Tbody, (T)TIME_STEP, ld_AB);
	}
}

// mode CPU = 0, GPU = 1
template <typename T, bool MODE = 0>
__host__
void testDynGrad(){
	// allocate
	T *h_x, *h_u, *h_AB, *h_AB2;
	int ld_AB = DIM_AB_r;	int ld_x = DIM_x_r;		int ld_u = DIM_u_r;
	h_x = (T *)malloc(ld_x*NUM_TIME_STEPS*sizeof(T));
	h_u = (T *)malloc(ld_u*NUM_TIME_STEPS*sizeof(T));
	h_AB = (T *)malloc(ld_AB*DIM_AB_c*NUM_TIME_STEPS*sizeof(T));
	h_AB2 = (T *)malloc(ld_AB*DIM_AB_c*NUM_TIME_STEPS*sizeof(T));
	#if MODE
		T *d_x, *d_u, *d_AB, *d_AB2;
		gpuErrchk(hipMalloc((void**)&d_x,ld_x*NUM_TIME_STEPS*sizeof(T)));
		gpuErrchk(hipMalloc((void**)&d_u,ld_u*NUM_TIME_STEPS*sizeof(T)));
		gpuErrchk(hipMalloc((void**)&d_AB,ld_AB*DIM_AB_c*NUM_TIME_STEPS*sizeof(T)));
		gpuErrchk(hipMalloc((void**)&d_AB2,ld_AB*DIM_AB_c*NUM_TIME_STEPS*sizeof(T)));
	#endif

	// load random states and controls
	for (int k=0; k < NUM_TIME_STEPS; k++){
		T *xk = &h_x[ld_x*k];	T *uk = &h_u[ld_u*k];
		for (int i=0; i < NUM_POS; i++){
			xk[i] = static_cast<T>(randDistq(randEng));
			xk[i+NUM_POS] = static_cast<T>(randDistqd(randEng));
			uk[i] = static_cast<T>(randDistu(randEng));	
		}
	}
	#if MODE
		gpuErrchk(hipMemcpy(d_x,h_x,ld_x*NUM_TIME_STEPS*sizeof(T),hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(d_u,h_u,ld_u*NUM_TIME_STEPS*sizeof(T),hipMemcpyHostToDevice));
	#endif

	// allocate and load I and Tbody
	T *h_I, *h_Tbody;
	h_I = (T *)malloc(36*NUM_POS*sizeof(T));	
	h_Tbody = (T *)malloc(36*NUM_POS*sizeof(T));	
	initI<T>(h_I);	initT<T>(h_Tbody);
	#if MODE
		T *d_I, *d_Tbody;
		gpuErrchk(hipMalloc((void**)&d_I,36*NUM_POS*sizeof(T)));	
		gpuErrchk(hipMalloc((void**)&d_Tbody,36*NUM_POS*sizeof(T)));
		gpuErrchk(hipMemcpy(d_I, h_I, 36*NUM_POS*sizeof(T), hipMemcpyHostToDevice));
		gpuErrchk(hipMemcpy(d_Tbody, h_Tbody, 36*NUM_POS*sizeof(T), hipMemcpyHostToDevice));
	#endif

	// compute grads
	#if MODE
		dim3 aGrid(NUM_TIME_STEPS,1);
		dim3 fdGrid(1,1);//NUM_TIME_STEPS,STATE_SIZE_PDDP+CONTROL_SIZE);
		dim3 aThreads(8,7);
		dim3 fdThreads(8,7);
		integratorGradientKernAnalytic<T><<<aGrid,aThreads>>>(d_AB,d_x,d_u,d_I,d_Tbody,ld_x,ld_u,ld_AB);
		integratorGradientKernFiniteDiff<T><<<fdGrid,fdThreads>>>(d_AB2,d_x,d_u,d_I,d_Tbody,ld_x,ld_u,ld_AB);
		gpuErrchk(hipDeviceSynchronize());
		gpuErrchk(hipMemcpy(h_AB,d_AB,ld_AB*DIM_AB_c*NUM_TIME_STEPS*sizeof(T),hipMemcpyDeviceToHost));
		gpuErrchk(hipMemcpy(h_AB2,d_AB2,ld_AB*DIM_AB_c*NUM_TIME_STEPS*sizeof(T),hipMemcpyDeviceToHost));
	#else
		integratorGradientAnalytic<T>(h_AB,h_x,h_u,h_I,h_Tbody,ld_x,ld_u,ld_AB);
		integratorGradientFiniteDiff<T>(h_AB2,h_x,h_u,h_I,h_Tbody,ld_x,ld_u,ld_AB);
	#endif

	// compare and print
	for (int k=0; k < NUM_TIME_STEPS-1; k++){
		T *ABk = &h_AB[ld_AB*DIM_AB_c*k];
		T *AB2k = &h_AB2[ld_AB*DIM_AB_c*k];
		for (int c=0; c < DIM_AB_c; c++){
			for (int r=0; r < DIM_AB_r; r++){
				int ind = c*ld_AB + r;
				T err = abs((ABk[ind] - AB2k[ind])/(ABk[ind] != 0 ? ABk[ind] : (AB2k[ind] != 0 ? AB2k[ind] : 1)));
				if (err > ERR_TOL){
					printf("k[%d] c,r[%d,%d]=ind[%d] has err[%.8f] for analytical[%.8f] vs finiteDiff[%.8f]\n",k,c,r,ind,err,ABk[ind],AB2k[ind]);
				}
			}
		}
	}

	//free
	free(h_x);		free(h_u);		free(h_AB);		free(h_I);		free(h_Tbody);
	#if MODE
		hipFree(d_x);	hipFree(d_u);	hipFree(d_AB);	hipFree(d_I);	hipFree(d_Tbody);
	#endif
}

char errMsg[]  = "Error: Unkown code - usage is [C]PU or [G]PU\n";
int main(int argc, char *argv[])
{
	srand(time(NULL));
	char hardware = '?'; // require user input
	if (argc > 1){hardware = argv[1][0];}
	if (hardware == 'C'){testDynGrad<algType,0>();}
	else if (hardware == 'G'){testDynGrad<algType,1>();}
	else{printf("%s",errMsg); hardware = '?';}
	return 0;
}